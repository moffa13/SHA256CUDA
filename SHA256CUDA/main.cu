#include "hip/hip_runtime.h"



#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <chrono>
#include <cmath>
#include <thread>
#include <iomanip>
#include <string>
#include <cassert>
#include <cstring>
#include<sstream>

#include "main.h"
#include "sha256.cuh"

#define SHOW_INTERVAL_MS 2000
#define ITERATIONS_PER_KERNEL 256
#define BLOCK_SIZE 256
#define NUMBLOCKS 163834u
#define IDX_MULTIPLIER (NUMBLOCKS * BLOCK_SIZE * ITERATIONS_PER_KERNEL)

static size_t difficulty = 1;

// Output string by the device read by host
char* g_out = nullptr;
unsigned char* g_hash_out = nullptr;
int* g_found = nullptr;

static uint64_t nonce_low = 0;
static uint64_t nonce_high = 0;
static uint64_t user_nonce_low = 0;
static uint64_t user_nonce_high = 0;
static uint64_t last_nonce_since_update = 0;

// Last timestamp we printed debug info
static std::chrono::high_resolution_clock::time_point t_last_updated;

__device__ bool checkZeroPadding(unsigned char* sha, uint8_t difficulty) {
    uint8_t fullZeros = difficulty / 2;
    uint8_t remainder = difficulty % 2;

    // Check full zero bytes first
    for (uint8_t i = 0; i < fullZeros; i++) {
        if (sha[i] != 0) return false;
    }

    // Check the last byte based on remainder
    if (remainder) {
        if (sha[fullZeros] == 0 || sha[fullZeros] > 0x0F) return false;
    }
    else {
        if (sha[fullZeros] <= 0x0F) return false;
    }

    return true;
}

__device__ void fast_memcpy_8bytes(unsigned char* dst, unsigned char* src) {
    dst[0] = src[0];
    dst[1] = src[1];
    dst[2] = src[2];
    dst[3] = src[3];
    dst[4] = src[4];
    dst[5] = src[5];
    dst[6] = src[6];
    dst[7] = src[7];
}

__device__ void nonce_to_bytes(uint64_t nonce_low, uint64_t nonce_high, unsigned char* out) {
    fast_memcpy_8bytes(out, (unsigned char*)&nonce_low);
    fast_memcpy_8bytes(out + 8, (unsigned char*)&nonce_high);
}

__constant__ uint64_t total_nonces = IDX_MULTIPLIER;
__constant__ unsigned char constant_bytes[4] = { 0xD8, 0x79, 0x9f, 0x50 };
__global__ void sha256_kernel(uint64_t* out_nonce_low, uint64_t* out_nonce_high, unsigned char* out_found_hash, int* out_found, const char* in_input_string, size_t in_input_string_size, uint8_t difficulty, uint64_t nonce_offset_low, uint64_t nonce_offset_high) {
    __shared__ SHA256_CTX shared_ctx[BLOCK_SIZE];
    uint64_t nonce_low;
    uint64_t nonce_high;

    unsigned char sha[32];

    for (uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        idx < total_nonces;
        idx += blockDim.x * gridDim.x) {
        nonce_low = idx + nonce_offset_low;
        nonce_high = nonce_offset_high;

        // Handle overflow. If nonce_low overflows, increment nonce_high.
        if (nonce_low < idx) {
            nonce_high++;
        }

        // Initialize SHA256 context
        sha256_init(&shared_ctx[threadIdx.x]);

        // Directly write constant_bytes and nonce to ctx.data
        shared_ctx[threadIdx.x].data[0] = constant_bytes[0];
        shared_ctx[threadIdx.x].data[1] = constant_bytes[1];
        shared_ctx[threadIdx.x].data[2] = constant_bytes[2];
        shared_ctx[threadIdx.x].data[3] = constant_bytes[3];
        nonce_to_bytes(nonce_low, nonce_high, shared_ctx[threadIdx.x].data + 4);

        // Adjust data length accordingly
        shared_ctx[threadIdx.x].datalen = 20; // 4 bytes for constant_bytes + 16 bytes for nonce

        sha256_update(&shared_ctx[threadIdx.x], (unsigned char*)in_input_string, in_input_string_size);
        sha256_final(&shared_ctx[threadIdx.x], sha);

        sha256_init(&shared_ctx[threadIdx.x]);
        sha256_update(&shared_ctx[threadIdx.x], sha, 32);
        sha256_final(&shared_ctx[threadIdx.x], sha);

        if (checkZeroPadding(sha, difficulty) && atomicExch(out_found, 1) == 0) {
            memcpy(out_found_hash, sha, 32);
            *out_nonce_low = nonce_low;
            *out_nonce_high = nonce_high;
        }
    }
}

void pre_sha256() {
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}

void print_hex(char* someHexData, size_t length) {
    for (size_t i = 0; i < length; i++) {
        printf("%02x", someHexData[i]);
    }
}

void print_hex(unsigned char* someHexData, size_t length) {
    for (size_t i = 0; i < length; i++) {
        printf("%02x", someHexData[i]);
    }
}

void print_state() {
    std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double, std::milli> last_show_interval = t2 - t_last_updated;

    if (last_show_interval.count() > SHOW_INTERVAL_MS) {
        std::chrono::duration<double, std::milli> span = t2 - t_last_updated;
        float ratio = span.count() / 1000;

        std::cout << std::fixed << static_cast<uint64_t>((nonce_low - last_nonce_since_update) / ratio) << " hash(es)/s" << std::endl;


        std::cout << std::fixed << "Nonce : " << nonce_low << std::endl;

        t_last_updated = std::chrono::high_resolution_clock::now();
        last_nonce_since_update = nonce_low;
    }
}

void hex_to_u64s(const std::string& hex, uint64_t& high, uint64_t& low) {

    std::stringstream ss;

    ss << std::hex << hex.substr(0, 16); // First 16 characters
    ss >> high;
    ss.clear();
    ss << std::hex << hex.substr(16, 16); // Last 16 characters
    ss >> low;
}

void hex_string_to_bytes(const char* hex_input, char* byte_output, size_t input_size) {
    for (size_t i = 0; i < input_size; i += 2) {
        // Take two characters from the hex string at a time
        char hex_byte[3];
        hex_byte[0] = hex_input[i];
        hex_byte[1] = hex_input[i + 1];
        hex_byte[2] = '\0';  // Null-terminate the string

        // Convert the two-character hex string to a byte
        byte_output[i / 2] = (char)strtol(hex_byte, NULL, 16);
    }
}

void to_hex_string(uint64_t a, uint64_t b, char* output) {
    unsigned char bytes[16];

    // Extract bytes
    for (int i = 0; i < 8; i++) {
        bytes[i] = (a >> (8 * (7 - i))) & 0xFF;
        bytes[8 + i] = (b >> (8 * (7 - i))) & 0xFF;
    }

    // Convert bytes to hex
    for (int i = 0; i < 16; i++) {
        sprintf(output + 2 * i, "%02x", bytes[i]);
    }
}

int main(int argc, char* argv[]) {
    if (argc < 3) {
        std::cerr << "Usage: <program> <message> <nonce hex> <difficulty>" << std::endl;
        return 1;
    }

    hipSetDevice(0);
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);

    t_last_updated = std::chrono::high_resolution_clock::now();

    std::string in, user_nonce_hex;

    in = argv[1];
    user_nonce_hex = argv[2];

    hex_to_u64s(user_nonce_hex, user_nonce_low, user_nonce_high);

    difficulty = std::stoul(argv[3]);

    // hex decode user input "string"
    size_t byte_array_size = in.size() / 2;
    char* byte_array = new char[byte_array_size];
    hex_string_to_bytes(in.c_str(), byte_array, in.size());

    // Input string for the device
    char* d_in = nullptr;

    // Create the input string for the device
    hipMalloc(&d_in, byte_array_size);
    hipMemcpy(d_in, byte_array, byte_array_size, hipMemcpyHostToDevice);

    uint64_t* g_nonce_low;
    uint64_t* g_nonce_high;
    hipMallocManaged((void**)&g_nonce_low, sizeof(uint64_t));
    hipMallocManaged((void**)&g_nonce_high, sizeof(uint64_t));

    hipMallocManaged(&g_hash_out, 32);
    hipMallocManaged(&g_found, sizeof(int));
    *g_found = 0;


    nonce_low += user_nonce_low;
    nonce_high += user_nonce_high;


    pre_sha256();


    while (!*g_found) {
        sha256_kernel << < NUMBLOCKS, BLOCK_SIZE >> > (g_nonce_low, g_nonce_high, g_hash_out, g_found, d_in, byte_array_size, difficulty, nonce_low, nonce_high);

        hipError_t err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            throw std::runtime_error("Device error");
        }
        nonce_low += IDX_MULTIPLIER;
        if (nonce_low < IDX_MULTIPLIER) {
            nonce_high++;
        }
        //print_state();
    }


    char hex_output[33];
    to_hex_string(*g_nonce_low, *g_nonce_high, hex_output);

    print_hex(g_hash_out, 32);
    printf("|");
    printf("%s", hex_output);


    hipFree(g_out);
    hipFree(g_hash_out);
    hipFree(g_found);

    hipFree(d_in);
    delete[] byte_array;
    hipDeviceReset();

    return 0;
}